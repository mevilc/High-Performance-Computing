
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 32

__global__ void MatrixMultiply(float *d_A, float *d_B, float *d_C)
{
	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

    for(int k=0; k<N; k++){
        d_C[row * N + col] += d_A[row * N + k] * d_B[k * N + col];	
    }
}

int main(){
	
	float h_A[N * N], h_B[N*N], h_C[N*N]; // host matrices

	// size to allocate
	size_t size = N*N*sizeof(float);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    //Initialize matrices on the host
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			h_A[i*N+j]=i;
			h_B[i*N+j]=i+1;
	    }
	}

	//Allocate Device memory
    float *d_A, *d_B, *d_C;	// devices matrices
	hipMalloc(&d_A, size);
	hipMalloc(&d_B, size);
	hipMalloc(&d_C, size);

	//Allocate A and B to the Device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	//Invoke kernel
	dim3 blockPerGrid(1,1);
	dim3 threadPerBlock(N,N); // thread.x = N, thread.y = N

	hipEventRecord(start, 0);	
	// Launch kernel
	MatrixMultiply<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop, 0);	
	//cudaEventSynchronize(stop);

	//Read C from device
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Elapsed time (ms): %f\n", milliseconds);
	
	// Check result on the CPU
	// For every row...
	for (int i = 0; i < N; i++) {
		// For every column...
		for (int j = 0; j < N; j++) {
			// For every element in the row-column pair
			int tmp = 0;
			for (int k = 0; k < N; k++) {
				// Accumulate the partial results
				tmp += h_A[i * N + k] * h_B[k * N + j];
			}

			// Check against the CPU result
			if(tmp != h_C[i * N + j]) {
				printf("Does not match!\n");
			}
		}
	}

	/*=============================Finish Test=================================*/

	//free(test_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(h_A);
	hipFree(h_B);
	hipFree(h_C);
	hipDeviceReset();
	return 0;
}